#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>
#include <random>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <sstream>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#ifndef M_E
#define M_E 2.71828182845904523536
#endif

using namespace std;

struct Agent {
    double *x;
    double B;
    double I;
    double objectivFunc;
};

__global__ void initializeCurand(hiprandState *states, int pop_size, unsigned long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < pop_size) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__device__ double sphereFunction(const double *x, int dim) {
    double sum = 0.0;
    for (int i = 0; i < dim; i++) {
        sum += x[i] * x[i];
    }
    return sum;
}

__global__ void evaluateFitness(Agent *fireflies, int pop_size, int dim) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < pop_size) {
        fireflies[idx].objectivFunc = sphereFunction(fireflies[idx].x, dim);
        fireflies[idx].I = 1.0 / fireflies[idx].objectivFunc;
    }
}

__global__ void updateFireflies(Agent *fireflies, int pop_size, int dim, double beta_base, double gamma, double alpha, double lb, double ub, hiprandState *states) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < pop_size) {
        hiprandState state = states[i];

        for (int j = 0; j < pop_size; j++) {
            double r = 0.0;
            for (int k = 0; k < dim; k++) {
                r += (fireflies[i].x[k] - fireflies[j].x[k]) * (fireflies[i].x[k] - fireflies[j].x[k]);
            }
            r = sqrt(r);

            if (fireflies[j].I > fireflies[i].I) {
                fireflies[i].B = beta_base * exp(-gamma * r * r);
                for (int k = 0; k < dim; k++) {
                    double u = alpha * (hiprand_uniform(&state) - 0.5);
                    fireflies[i].x[k] += fireflies[i].B * (fireflies[j].x[k] - fireflies[i].x[k]) + u;
                    fireflies[i].x[k] = max(lb, min(ub, fireflies[i].x[k]));
                }
            }
        }
        states[i] = state;
        __syncthreads();
    }
}

void generate_population(int pop_size, int dim, Agent *fireflies, double lb, double ub) {
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<double> dis(lb, ub);

    for (int i = 0; i < pop_size; i++) {
        fireflies[i].B = 1.0;
        hipMallocManaged(&fireflies[i].x, dim * sizeof(double));
        for (int j = 0; j < dim; j++) {
            fireflies[i].x[j] = dis(gen);
        }
    }
}

int main() {
    vector<int> dimensions = {10, 30, 50};
    vector<int> popSizes = {30, 50, 70};

    for (int dim : dimensions) {
        for (int popSize : popSizes) {
            cout << "dim: " << dim << " popSize: " << popSize << endl;
            stringstream ss;
            ss << "results_dim_" << dim << "_pop_" << popSize << ".txt";
            string fileName = ss.str();
            ofstream outFile(fileName);
            if (!outFile) {
                cerr << "Cannot open output file: " << fileName << endl;
                return 1;
            }

            int pop_size = popSize;
            int n_dim = dim;
            double lb = -10.0, ub = 10.0;
            double alpha = 0.2, beta_base = 2.0, gamma = 0.001;
            unsigned long long seed = 12345678;

            Agent *fireflies;
            hipMallocManaged(&fireflies, pop_size * sizeof(Agent));
            generate_population(pop_size, n_dim, fireflies, lb, ub);

            hiprandState *d_states;
            hipMalloc(&d_states, pop_size * sizeof(hiprandState));
            initializeCurand<<<(pop_size + 255) / 256, 256>>>(d_states, pop_size, seed);
            hipDeviceSynchronize();

            int blockSize = 256;
            int numBlocks = (pop_size + blockSize - 1) / blockSize;

            // Mesure du temps avec CUDA Events
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);

            for (int run = 0; run < 10; run++) {
                for (int t = 0; t < 5000; t++) {
                    updateFireflies<<<numBlocks, blockSize>>>(fireflies, pop_size, n_dim, beta_base, gamma, alpha, lb, ub, d_states);
                    evaluateFitness<<<numBlocks, blockSize>>>(fireflies, pop_size, n_dim);
                    hipDeviceSynchronize();
                }

                double bestFitness = fireflies[0].objectivFunc;
                for (int i = 1; i < pop_size; i++) {
                    if (fireflies[i].objectivFunc < bestFitness) {
                        bestFitness = fireflies[i].objectivFunc;
                    }
                }
                outFile << bestFitness << endl;
                printf("Run %d: Best Fitness = %lf\n", run, bestFitness);
            }

            // Fin de la mesure du temps
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Execution Time for dim=%d, popSize=%d: %f ms\n", dim, popSize, milliseconds);

            outFile.close();

            for (int i = 0; i < pop_size; i++) {
                hipFree(fireflies[i].x);
            }
            hipFree(fireflies);
            hipFree(d_states);

            // Libération des CUDA Events
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }

    return 0;
}
